#include "hip/hip_runtime.h"
#include "kan_cuda.h"
#include <hip/hip_runtime.h>

__global__ void kan_forward_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = x * x; // Placeholder for φ(x)
    }
}

void runKANForwardCUDA(const float* input, float* output, int size) {
    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    kan_forward_kernel<<<blocks, threads>>>(d_input, d_output, size);

    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
